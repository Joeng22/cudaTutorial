
#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>

__global__
void add(int n,float *x,float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
    //printf("blockIdx.x: %d, threadIdx.x: %d  blockDim.x=%d gridStride: %d \n", blockId, index,blockDim.x,gridStride);
}


int main()
{
    int N = 1<<20;
    float *x,*y;

    printf("N=%d\n",N);
    hipMallocManaged(&x,N*sizeof(float));
    hipMallocManaged(&y,N*sizeof(float)); 

    for(int i=0;i<N;i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }   
    int threadsPerBlock = 256;
    int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocksPerGrid,threadsPerBlock>>>(N,x,y);
    hipDeviceSynchronize();

    // Free memory
    hipFree(x);
    hipFree(y);
    
    return 0;
}